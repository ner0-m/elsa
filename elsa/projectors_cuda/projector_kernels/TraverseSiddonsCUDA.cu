#include "hip/hip_runtime.h"
#include "TraverseSiddonsCUDA.cuh"

constexpr uint32_t MAX_THREADS_PER_BLOCK =
    elsa::TraverseSiddonsCUDA<float, 3>::MAX_THREADS_PER_BLOCK;

template <typename data_t, uint32_t size>
struct EasyAccessSharedArray {
    data_t* const __restrict__ _p;

    __device__ EasyAccessSharedArray(data_t* p) : _p{p + threadIdx.x} {}

    __device__ __forceinline__ const data_t& operator[](uint32_t index) const
    {
        return _p[index * MAX_THREADS_PER_BLOCK];
    }

    __device__ __forceinline__ data_t& operator[](uint32_t index)
    {
        return _p[index * MAX_THREADS_PER_BLOCK];
    }
};
/**
 * \brief General square matrix-vector multiplication
 *
 * important: always use byte pointers for multidimensional arrays
 */
template <typename real_t, uint32_t dim>
__device__ __forceinline__ void
    gesqmv(const int8_t* const __restrict__ matrix, const real_t* const __restrict__ vector,
           real_t* const __restrict__ result, const uint32_t matrixPitch)
{
    // initialize result vector
    real_t* columnPtr = (real_t*) matrix;
#pragma unroll
    for (uint32_t x = 0; x < dim; x++) {
        result[x] = columnPtr[x] * vector[0];
    }

// accumulate results for remaning columns
#pragma unroll
    for (uint32_t y = 1; y < dim; y++) {
        real_t* columnPtr = (real_t*) (matrix + matrixPitch * y);
#pragma unroll
        for (uint32_t x = 0; x < dim; x++) {
            result[x] += columnPtr[x] * vector[y];
        }
    }
}

/// normalizes a vector of length 2 or 3 using device inbuilt functions
template <typename real_t, uint32_t dim>
__device__ __forceinline__ void normalize(real_t* const __restrict__ vector)
{
    real_t rn;
    if (dim == 3)
        rn = rnorm3d(vector[0], vector[1], vector[2]);
    else if (dim == 2)
        rn = rhypot(vector[0], vector[1]);

#pragma unroll
    for (int i = 0; i < dim; i++) {
        vector[i] *= rn;
    }
}

/// calculates the point at a distance delta from the ray origin ro in direction rd
template <typename real_t, uint32_t dim>
__device__ __forceinline__ void pointAt(const real_t* const __restrict__ ro,
                                        const real_t* const __restrict__ rd, const real_t& delta,
                                        real_t* const __restrict__ result)
{
#pragma unroll
    for (int i = 0; i < dim; i++)
        result[i] = delta * rd[i] + ro[i];
}

/// projects a point onto the bounding box by clipping (points inside the bounding box are
/// unaffected)
template <typename real_t, uint32_t dim>
__device__ __forceinline__ void projectOntoBox(real_t* const __restrict__ point,
                                               const EasyAccessSharedArray<uint32_t, dim>& boxMax)
{
#pragma unroll
    for (int i = 0; i < dim; i++) {
        point[i] = point[i] < 0.0f ? 0.0f : point[i];
        point[i] = point[i] > boxMax[i] ? boxMax[i] : point[i];
    }
}

/// determines the voxel that contains a point, if the point is on a border the voxel in the ray
/// direction is favored
template <typename real_t, uint32_t dim>
__device__ __forceinline__ bool closestVoxel(const real_t* const __restrict__ point,
                                             const EasyAccessSharedArray<uint32_t, dim>& boxMax,
                                             EasyAccessSharedArray<uint32_t, dim>& voxelCoord,
                                             const real_t* const __restrict__ rd,
                                             const EasyAccessSharedArray<int, dim>& stepDir)
{
#pragma unroll
    for (int i = 0; i < dim; i++) {
        // point has been projected onto box => point[i]>=0, can use uint32_t
        real_t fl = trunc(point[i]);
        voxelCoord[i] = fl == point[i] && rd[i] < 0.0f ? fl - 1.0f : fl;
        if (voxelCoord[i] >= boxMax[i])
            return false;
    }
    return true;
}

/// initializes stepDir with the sign of rd
template <typename real_t, uint32_t dim>
__device__ __forceinline__ void initStepDirection(const real_t* const __restrict__ rd,
                                                  EasyAccessSharedArray<int, dim>& stepDir)
{
#pragma unroll
    for (int i = 0; i < dim; i++)
        stepDir[i] = ((rd[i] > 0.0f) - (rd[i] < 0.0f));
}

/// initialize step sizes considering the ray direcion
template <typename real_t, uint32_t dim>
__device__ __forceinline__ void initDelta(const real_t* const __restrict__ rd,
                                          const EasyAccessSharedArray<int, dim>& stepDir,
                                          EasyAccessSharedArray<real_t, dim>& delta)
{
#pragma unroll
    for (int i = 0; i < dim; i++) {
        real_t d = stepDir[i] / rd[i];
        delta[i] = rd[i] >= -__FLT_EPSILON__ && rd[i] <= __FLT_EPSILON__ ? __FLT_MAX__ : d;
    }
}

/// initialize maximum step parameters considering the ray direction
template <typename real_t, uint32_t dim>
__device__ __forceinline__ void initMax(const real_t* const __restrict__ rd,
                                        const EasyAccessSharedArray<uint32_t, dim>& currentVoxel,
                                        const real_t* const __restrict__ point,
                                        EasyAccessSharedArray<real_t, dim>& tmax)
{
    real_t nextBoundary;
#pragma unroll
    for (int i = 0; i < dim; i++) {
        nextBoundary = rd[i] > 0.0f ? currentVoxel[i] + 1 : currentVoxel[i];
        tmax[i] = rd[i] >= -__FLT_EPSILON__ && rd[i] <= __FLT_EPSILON__
                      ? __FLT_MAX__
                      : (nextBoundary - point[i]) / rd[i];
    }
}

/// find intersection points of ray with AABB
template <typename real_t, uint32_t dim>
__device__ __forceinline__ bool
    box_intersect(const real_t* const __restrict__ ro, const real_t* const __restrict__ rd,
                  const EasyAccessSharedArray<uint32_t, dim>& boxMax, real_t& tmin)
{
    real_t invDir = 1.0f / rd[0];

    real_t t1 = -ro[0] * invDir;
    real_t t2 = (boxMax[0] - ro[0]) * invDir;

    /**
     * fmin and fmax adhere to the IEEE standard, and return the non-NaN element if only a single
     * NaN is present
     */
    // tmin and tmax have to be picked for each specific direction without using fmin/fmax
    // (supressing NaNs is bad in this case)
    tmin = invDir >= 0 ? t1 : t2;
    real_t tmax = invDir >= 0 ? t2 : t1;

#pragma unroll
    for (int i = 1; i < dim; ++i) {
        invDir = 1.0f / rd[i];

        t1 = -ro[i] * invDir;
        t2 = (boxMax[i] - ro[i]) * invDir;

        tmin = fmax(tmin, invDir >= 0 ? t1 : t2);
        tmax = fmin(tmax, invDir >= 0 ? t2 : t1);
    }

    if (tmax == 0.0f && tmin == 0.0f)
        return false;
    if (tmax >= fmax(tmin, 0.0f)) // hit
        return true;
    return false;
}

/// returns the index of the smallest element in an array
template <typename real_t, uint32_t dim>
__device__ __forceinline__ uint32_t minIndex(const EasyAccessSharedArray<real_t, dim>& tmax)
{
    uint32_t index = 0;
    real_t min = tmax[0];

#pragma unroll
    for (int i = 1; i < dim; i++) {
        bool cond = tmax[i] < min;
        index = cond ? i : index;
        min = cond ? tmax[i] : min;
    }

    return index;
}

/// checks whether the voxel lies inside the AABB
template <typename real_t, uint32_t dim>
__device__ __forceinline__ bool
    isVoxelInVolume(const EasyAccessSharedArray<uint32_t, dim>& currentVoxel,
                    const EasyAccessSharedArray<uint32_t, dim>& boxMax, const uint32_t& index)
{
    return currentVoxel[index] < boxMax[index];
}

/// updates the traversal algorithm, after update the current position will be the exit point from
/// current voxel
template <typename real_t, uint32_t dim>
__device__ __forceinline__ real_t updateTraverse(EasyAccessSharedArray<uint32_t, dim>& currentVoxel,
                                                 const EasyAccessSharedArray<int, dim>& stepDir,
                                                 const EasyAccessSharedArray<real_t, dim>& tdelta,
                                                 EasyAccessSharedArray<real_t, dim>& tmax,
                                                 real_t& texit, uint32_t& index)
{
    real_t tentry = texit;

    index = minIndex<real_t, dim>(tmax);
    texit = tmax[index];

    currentVoxel[index] += stepDir[index];
    tmax[index] += tdelta[index];

    return texit - tentry;
}

/*
 * atomicAdd() for doubles is only supported on devices of compute capability 6.0 or higher
 * implementation taken straight from the CUDA C programming guide:
 * https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#atomic-functions
 */
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600
__device__ __forceinline__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull = (unsigned long long int*) address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val + __longlong_as_double(assumed)));

        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

template <typename data_t, bool adjoint, uint32_t dim>
__global__ void __launch_bounds__(elsa::TraverseSiddonsCUDA<data_t, dim>::MAX_THREADS_PER_BLOCK)
    traverseVolume(int8_t* const __restrict__ volume, const uint64_t volumePitch,
                   int8_t* const __restrict__ sinogram, const uint64_t sinogramPitch,
                   const uint32_t sinogramOffsetX, const int8_t* const __restrict__ rayOrigins,
                   const uint32_t originPitch, const int8_t* const __restrict__ projInv,
                   const uint32_t projPitch,
                   const typename elsa::TraverseSiddonsCUDA<data_t, dim>::BoundingBox boundingBox)
{
    using real_t = elsa::real_t;

    const int8_t* const projInvPtr = projInv + blockIdx.x * projPitch * dim;

    const real_t* const rayOrigin = (real_t*) (rayOrigins + blockIdx.x * originPitch);

    const uint32_t xCoord = sinogramOffsetX + blockDim.x * blockIdx.z + threadIdx.x;

    data_t* sinogramPtr =
        ((data_t*) (sinogram + (blockIdx.x * gridDim.y + blockIdx.y) * sinogramPitch) + xCoord);

    // homogenous pixel coordinates
    real_t pixelCoord[dim];
    pixelCoord[0] = xCoord + 0.5f;
    pixelCoord[dim - 1] = 1.0f;
    if (dim == 3)
        pixelCoord[1] = blockIdx.y + 0.5f;

    __shared__ uint32_t currentVoxelsShared[MAX_THREADS_PER_BLOCK * dim];
    __shared__ int stepDirsShared[MAX_THREADS_PER_BLOCK * dim];
    __shared__ real_t tdeltasShared[MAX_THREADS_PER_BLOCK * dim];
    __shared__ real_t tmaxsShared[MAX_THREADS_PER_BLOCK * dim];
    __shared__ uint32_t boxMaxsShared[MAX_THREADS_PER_BLOCK * dim];

    EasyAccessSharedArray<uint32_t, dim> boxMax{boxMaxsShared};
#pragma unroll
    for (uint32_t i = 0; i < dim; i++)
        boxMax[i] = boundingBox[i];

    // compute ray direction
    real_t rd[dim];
    gesqmv<real_t, dim>(projInvPtr, pixelCoord, rd, projPitch);
    normalize<real_t, dim>(rd);

    // find volume intersections
    real_t tmin;
    if (!box_intersect<real_t, dim>(rayOrigin, rd, boxMax, tmin))
        return;

    real_t entryPoint[dim];
    pointAt<real_t, dim>(rayOrigin, rd, tmin, entryPoint);
    projectOntoBox<real_t, dim>(entryPoint, boxMax);

    EasyAccessSharedArray<int, dim> stepDir{stepDirsShared};
    initStepDirection<real_t, dim>(rd, stepDir);

    EasyAccessSharedArray<uint32_t, dim> currentVoxel{currentVoxelsShared};
    if (!closestVoxel<real_t, dim>(entryPoint, boxMax, currentVoxel, rd, stepDir))
        return;

    EasyAccessSharedArray<real_t, dim> tdelta{tdeltasShared};
    EasyAccessSharedArray<real_t, dim> tmax{tmaxsShared};
    initDelta<real_t, dim>(rd, stepDir, tdelta);
    initMax<real_t, dim>(rd, currentVoxel, entryPoint, tmax);

    uint32_t index;
    real_t texit = 0.0f;
    real_t pixelValue = 0.0f;

    data_t* volumeXPtr =
        dim == 3 ? (data_t*) (volume
                              + (boundingBox[1] * currentVoxel[2] + currentVoxel[1]) * volumePitch)
                       + currentVoxel[0]
                 : (data_t*) (volume + currentVoxel[1] * volumePitch) + currentVoxel[0];
    do {
        real_t d = updateTraverse<real_t, dim>(currentVoxel, stepDir, tdelta, tmax, texit, index);
        if (adjoint)
            atomicAdd(volumeXPtr, *sinogramPtr * d);
        else
            pixelValue += d * (*volumeXPtr);

        volumeXPtr =
            dim == 3
                ? (data_t*) (volume
                             + (boundingBox[1] * currentVoxel[2] + currentVoxel[1]) * volumePitch)
                      + currentVoxel[0]
                : (data_t*) (volume + currentVoxel[1] * volumePitch) + currentVoxel[0];
    } while (isVoxelInVolume<real_t, dim>(currentVoxel, boxMax, index));

    if (!adjoint)
        *sinogramPtr = pixelValue;
}

namespace elsa
{
    template <typename data_t, uint32_t dim>
    void TraverseSiddonsCUDA<data_t, dim>::traverseForward(
        const dim3 sinogramDims, const int threads, int8_t* const __restrict__ volume,
        const uint64_t volumePitch, int8_t* const __restrict__ sinogram,
        const uint64_t sinogramPitch, const int8_t* const __restrict__ rayOrigins,
        const uint32_t originPitch, const int8_t* const __restrict__ projInv,
        const uint32_t projPitch, const BoundingBox& boundingBox)
    {
        uint32_t numImgBlocks = sinogramDims.z / threads;
        uint32_t remaining = sinogramDims.z % threads;
        uint32_t offset = numImgBlocks * threads;

        if (numImgBlocks > 0) {
            const dim3 grid(sinogramDims.x, sinogramDims.y, numImgBlocks);
            traverseVolume<data_t, false, dim>
                <<<grid, threads>>>(volume, volumePitch, sinogram, sinogramPitch, 0, rayOrigins,
                                    originPitch, projInv, projPitch, boundingBox);
        }

        if (remaining > 0) {
            hipStream_t remStream;
            if (hipStreamCreate(&remStream) != hipSuccess)
                throw std::logic_error(
                    "TraverseSiddonsCUDA: Couldn't create stream for remaining images");

            const dim3 grid(sinogramDims.x, sinogramDims.y, 1);
            traverseVolume<data_t, false, dim><<<grid, remaining, 0, remStream>>>(
                volume, volumePitch, sinogram, sinogramPitch, offset, rayOrigins, originPitch,
                projInv, projPitch, boundingBox);

            if (hipStreamDestroy(remStream) != hipSuccess)
                throw std::logic_error("TraverseSiddonsCUDA: Couldn't destroy GPU stream; This may "
                                       "cause problems later.");
        }
    }

    template <typename data_t, uint32_t dim>
    void TraverseSiddonsCUDA<data_t, dim>::traverseAdjoint(
        const dim3 sinogramDims, const int threads, int8_t* const __restrict__ volume,
        const uint64_t volumePitch, int8_t* const __restrict__ sinogram,
        const uint64_t sinogramPitch, const int8_t* const __restrict__ rayOrigins,
        const uint32_t originPitch, const int8_t* const __restrict__ projInv,
        const uint32_t projPitch, const BoundingBox& boundingBox)
    {
        uint32_t numImgBlocks = sinogramDims.z / threads;
        uint32_t remaining = sinogramDims.z % threads;
        uint32_t offset = numImgBlocks * threads;

        if (numImgBlocks > 0) {
            const dim3 grid(sinogramDims.x, sinogramDims.y, numImgBlocks);
            traverseVolume<data_t, true, dim>
                <<<grid, threads>>>(volume, volumePitch, sinogram, sinogramPitch, 0, rayOrigins,
                                    originPitch, projInv, projPitch, boundingBox);
        }

        if (remaining > 0) {
            hipStream_t remStream;
            if (hipStreamCreate(&remStream) != hipSuccess)
                throw std::logic_error(
                    "TraverseSiddonsCUDA: Couldn't create stream for remaining images");

            const dim3 grid(sinogramDims.x, sinogramDims.y, 1);
            traverseVolume<data_t, true, dim><<<grid, remaining, 0, remStream>>>(
                volume, volumePitch, sinogram, sinogramPitch, offset, rayOrigins, originPitch,
                projInv, projPitch, boundingBox);

            if (hipStreamDestroy(remStream) != hipSuccess)
                throw std::logic_error("TraverseSiddonsCUDA: Couldn't destroy GPU stream; This may "
                                       "cause problems later.");
        }
    }

    // ------------------------------------------
    // explicit template instantiation
    template struct TraverseSiddonsCUDA<float, 2>;
    template struct TraverseSiddonsCUDA<float, 3>;
    template struct TraverseSiddonsCUDA<double, 2>;
    template struct TraverseSiddonsCUDA<double, 3>;
} // namespace elsa