#include "hip/hip_runtime.h"
#include "TraverseJosephsCUDA.cuh"

/**
 * \brief General square matrix-vector multiplication
 *
 * important: always use byte pointers for multidimensional arrays
 */
template <typename real_t, uint32_t dim>
__device__ __forceinline__ void
    gesqmv(const int8_t* const __restrict__ matrix, const real_t* const __restrict__ vector,
           real_t* const __restrict__ result, const uint32_t matrixPitch)
{
    // initialize result vector
    real_t* columnPtr = (real_t*) matrix;
#pragma unroll
    for (uint32_t x = 0; x < dim; x++) {
        result[x] = columnPtr[x] * vector[0];
    }

// accumulate results for remaning columns
#pragma unroll
    for (uint32_t y = 1; y < dim; y++) {
        real_t* columnPtr = (real_t*) (matrix + matrixPitch * y);
#pragma unroll
        for (uint32_t x = 0; x < dim; x++) {
            result[x] += columnPtr[x] * vector[y];
        }
    }
}

/// determine reverse norm of vector of length 2 or 3 using device inbuilt functions
template <typename real_t, uint32_t dim>
__device__ __forceinline__ real_t rnorm(real_t* const __restrict__ vector)
{
    if (dim == 3)
        return rnorm3d(vector[0], vector[1], vector[2]);
    else if (dim == 2)
        return rhypot(vector[0], vector[1]);
    else
        return -1.0;
}

/// normalizes a vector of length 2 or 3 using device inbuilt norm
template <typename real_t, uint32_t dim>
__device__ __forceinline__ void normalize(real_t* const __restrict__ vector)
{
    real_t rn = rnorm<real_t, dim>(vector);

#pragma unroll
    for (int i = 0; i < dim; i++) {
        vector[i] *= rn;
    }
}

/// calculates the point at a distance delta from the ray origin ro in direction rd
template <typename real_t, uint32_t dim>
__device__ __forceinline__ void pointAt(const real_t* const __restrict__ ro,
                                        const real_t* const __restrict__ rd, const real_t delta,
                                        real_t* const __restrict__ result)
{
#pragma unroll
    for (int i = 0; i < dim; i++)
        result[i] = delta * rd[i] + ro[i];
}

/// projects a point onto the bounding box by clipping (points inside the bounding box are
/// unaffected)
template <typename real_t, uint32_t dim>
__device__ __forceinline__ void projectOntoBox(real_t* const __restrict__ point,
                                               const real_t* const __restrict__ boxMax)
{
#pragma unroll
    for (int i = 0; i < dim; i++) {
        point[i] = point[i] < 0.0f ? 0.0f : point[i];
        point[i] = point[i] > boxMax[i] ? boxMax[i] : point[i];
    }
}

/// determines the voxel that contains a point, if the point is on a border the voxel in the ray
/// direction is favored
template <typename real_t, uint32_t dim>
__device__ __forceinline__ bool
    closestVoxel(const real_t* const __restrict__ point, const real_t* const __restrict__ boxMax,
                 uint32_t* const __restrict__ voxelCoord, const real_t* const __restrict__ rd)
{
#pragma unroll
    for (int i = 0; i < dim; i++) {
        // point has been projected onto box => point[i]>=0, can use uint32_t
        uint32_t fl = trunc(point[i]);
        // for Joseph's also consider rays running along the "left" boundary
        voxelCoord[i] = fl == point[i] && rd[i] <= 0.0f && point[i] != 0.0f ? fl - 1 : fl;
        if (voxelCoord[i] >= boxMax[i])
            return false;
    }
    return true;
}

/// initializes stepDir with the sign of rd
template <typename real_t, uint32_t dim>
__device__ __forceinline__ void initStepDirection(const real_t* const __restrict__ rd,
                                                  int* const __restrict__ stepDir)
{
#pragma unroll
    for (int i = 0; i < dim; i++)
        stepDir[i] = ((rd[i] > 0.0f) - (rd[i] < 0.0f));
}

/// initialize step sizes considering the ray direcion
template <typename real_t, uint32_t dim>
__device__ __forceinline__ void initDelta(const real_t* const __restrict__ rd,
                                          const int* const __restrict__ stepDir,
                                          real_t* const __restrict__ delta)
{
#pragma unroll
    for (int i = 0; i < dim; i++) {
        real_t d = stepDir[i] / rd[i];
        delta[i] = rd[i] >= -__FLT_EPSILON__ && rd[i] <= __FLT_EPSILON__ ? __FLT_MAX__ : d;
    }
}

/// find intersection points of ray with AABB
template <typename real_t, uint32_t dim>
__device__ __forceinline__ bool
    box_intersect(const real_t* const __restrict__ ro, const real_t* const __restrict__ rd,
                  const real_t* const __restrict__ boxMax, real_t& tmin, real_t& tmax)
{
    real_t invDir = 1.0f / rd[0];

    real_t t1 = -ro[0] * invDir;
    real_t t2 = (boxMax[0] - ro[0]) * invDir;

    /**
     * fminf and fmaxf adhere to the IEEE standard, and return the non-NaN element if only a single
     * NaN is present
     */
    // tmin and tmax have to be picked for each specific direction without using fmin/fmax
    // (supressing NaNs is bad in this case)
    tmin = invDir >= 0 ? t1 : t2;
    tmax = invDir >= 0 ? t2 : t1;

#pragma unroll
    for (int i = 1; i < dim; ++i) {
        invDir = 1.0f / rd[i];

        t1 = -ro[i] * invDir;
        t2 = (boxMax[i] - ro[i]) * invDir;

        tmin = fmax(tmin, invDir >= 0 ? t1 : t2);
        tmax = fmin(tmax, invDir >= 0 ? t2 : t1);
    }

    if (tmax == 0.0f && tmin == 0.0f)
        return false;
    if (tmax >= fmax(tmin, 0.0f)) // hit
        return true;
    return false;
}

/// returns the index of the smallest element in an array
template <typename real_t, uint32_t dim>
__device__ __forceinline__ uint32_t minIndex(const real_t* const __restrict__ array)
{
    uint32_t index = 0;
    real_t min = array[0];

#pragma unroll
    for (int i = 1; i < dim; i++) {
        bool cond = array[i] < min;
        index = cond ? i : index;
        min = cond ? array[i] : min;
    }

    return index;
}

/// return the index of the element with the maximum absolute value in array
template <typename real_t, uint32_t dim>
__device__ __forceinline__ uint32_t maxAbsIndex(const real_t* const __restrict__ array)
{
    uint32_t index = 0;
    real_t max = fabs(array[0]);

#pragma unroll
    for (int i = 1; i < dim; i++) {
        bool cond = fabs(array[i]) > max;
        index = cond ? i : index;
        max = cond ? fabs(array[i]) : max;
    }

    return index;
}

/// currentPosition is advanced by dist in direction rd
template <typename real_t, uint32_t dim>
__device__ __forceinline__ void updateTraverse(real_t* const __restrict__ currentPosition,
                                               const real_t* const __restrict__ rd,
                                               const real_t dist)
{
#pragma unroll
    for (uint32_t i = 0; i < dim; i++)
        currentPosition[i] += rd[i] * dist;
}

/// convenience function for texture fetching
template <typename data_t, uint dim>
__device__ __forceinline__ data_t tex(hipTextureObject_t texObj, const elsa::real_t* const p)
{
    if (dim == 3)
        return tex3D<data_t>(texObj, p[0], p[1], p[2]);
    else
        return tex2D<data_t>(texObj, p[0], p[1]);
}

/// fetches double at position (x,y) from 2D texture
__device__ __forceinline__ double tex2Dd(hipTextureObject_t texObj, const float x, const float y)
{
    uint2 rt = tex2D<uint2>(texObj, x, y);
    return __hiloint2double(rt.y, rt.x);
}

/// template specialization for double texture fetches
template <>
__device__ __forceinline__ double tex<double, 2>(hipTextureObject_t texObj,
                                                 const elsa::real_t* const p)
{
    elsa::real_t x = p[0] - 0.5f;
    elsa::real_t y = p[1] - 0.5f;

    elsa::real_t i = floor(x);
    elsa::real_t j = floor(y);

    elsa::real_t a = x - i;
    elsa::real_t b = y - j;

    double T[2][2];
    T[0][0] = tex2Dd(texObj, i, j);
    T[1][0] = tex2Dd(texObj, i + 1, j);
    T[0][1] = tex2Dd(texObj, i, j + 1);
    T[1][1] = tex2Dd(texObj, i + 1, j + 1);

    return (1 - a) * (1 - b) * T[0][0] + a * (1 - b) * T[1][0] + (1 - a) * b * T[0][1]
           + a * b * T[1][1];
}

/// fetches double at position (x,y,z) from 3D texture
__device__ __forceinline__ double tex3Dd(hipTextureObject_t texObj, const float x, const float y,
                                         const elsa::real_t z)
{
    uint2 rt = tex3D<uint2>(texObj, x, y, z);
    return __hiloint2double(rt.y, rt.x);
}

/// template specialization for double texture fetches
template <>
__device__ __forceinline__ double tex<double, 3>(hipTextureObject_t texObj,
                                                 const elsa::real_t* const p)
{
    elsa::real_t x = p[0] - 0.5f;
    elsa::real_t y = p[1] - 0.5f;
    elsa::real_t z = p[2] - 0.5f;

    elsa::real_t i = floor(x);
    elsa::real_t j = floor(y);
    elsa::real_t k = floor(z);

    elsa::real_t a = x - i;
    elsa::real_t b = y - j;
    elsa::real_t c = z - k;

    double T[2][2][2];
    T[0][0][0] = tex3Dd(texObj, i, j, k);
    T[1][0][0] = tex3Dd(texObj, i + 1, j, k);
    T[0][1][0] = tex3Dd(texObj, i, j + 1, k);
    T[0][0][1] = tex3Dd(texObj, i, j, k + 1);
    T[1][1][0] = tex3Dd(texObj, i + 1, j + 1, k);
    T[1][0][1] = tex3Dd(texObj, i + 1, j, k + 1);
    T[0][1][1] = tex3Dd(texObj, i, j + 1, k + 1);
    T[1][1][1] = tex3Dd(texObj, i + 1, j + 1, k + 1);

    return (1 - a) * (1 - b) * (1 - c) * T[0][0][0] + a * (1 - b) * (1 - c) * T[1][0][0] +

           (1 - a) * b * (1 - c) * T[0][1][0] + a * b * (1 - c) * T[1][1][0] +

           (1 - a) * (1 - b) * c * T[0][0][1] + a * (1 - b) * c * T[1][0][1] +

           (1 - a) * b * c * T[0][1][1] + a * b * c * T[1][1][1];
}

template <typename data_t, uint dim>
__global__ void __launch_bounds__(elsa::TraverseJosephsCUDA<data_t, dim>::MAX_THREADS_PER_BLOCK)
    traverseForwardKernel(hipTextureObject_t volume, int8_t* const __restrict__ sinogram,
                          const uint64_t sinogramPitch, const int8_t* const __restrict__ rayOrigins,
                          const uint32_t originPitch, const int8_t* const __restrict__ projInv,
                          const uint32_t projPitch,
                          const typename elsa::TraverseJosephsCUDA<data_t, dim>::BoundingBox boxMax)
{

    using real_t = elsa::real_t;

    const int8_t* const projInvPtr =
        dim == 3 ? projInv + (blockIdx.z * blockDim.x + threadIdx.x) * projPitch * 3
                 : projInv + (blockIdx.y * blockDim.x + threadIdx.x) * projPitch * 2;

    const real_t* const rayOrigin =
        dim == 3 ? (real_t*) (rayOrigins + (blockIdx.z * blockDim.x + threadIdx.x) * originPitch)
                 : (real_t*) (rayOrigins + (blockIdx.y * blockDim.x + threadIdx.x) * originPitch);

    data_t* sinogramPtr =
        dim == 3 ? ((data_t*) (sinogram
                               + ((blockIdx.z * blockDim.x + threadIdx.x) * gridDim.y + blockIdx.y)
                                     * sinogramPitch)
                    + blockIdx.x)
                 : ((data_t*) (sinogram + (blockIdx.y * blockDim.x + threadIdx.x) * sinogramPitch)
                    + blockIdx.x);

    *sinogramPtr = 0;

    // homogenous pixel coordinates
    real_t pixelCoord[dim];
    pixelCoord[0] = blockIdx.x + 0.5f;
    pixelCoord[dim - 1] = 1.0f;
    if (dim == 3)
        pixelCoord[dim - 2] = blockIdx.y + 0.5f;

    // compute ray direction
    real_t rd[dim];
    gesqmv<real_t, dim>(projInvPtr, pixelCoord, rd, projPitch);

    // determine main direction
    const uint32_t idx = maxAbsIndex<real_t, dim>(rd);
    const real_t rdMax = abs(rd[idx]);

    real_t rn = rnorm<real_t, dim>(rd);

    real_t weight = rn / rdMax;

// normalize ray direction to have length 1/-1 in the main direction
#pragma unroll
    for (int i = 0; i < dim; ++i)
        rd[i] /= rdMax;

    // find volume intersections
    real_t tmin, tmax;
    if (!box_intersect<real_t, dim>(rayOrigin, rd, boxMax.max, tmin, tmax))
        return;

    real_t currentPosition[dim];
    pointAt<real_t, dim>(rayOrigin, rd, tmin, currentPosition);

    // truncate as currentPosition is non-negative
    const real_t fl = trunc(currentPosition[idx]);
    // for Joseph's also consider rays running along the "left" boundary
    const real_t firstBoundary = fl == currentPosition[idx] && rd[idx] < 0.0f ? fl - 1.0f : fl;

    // find distance to next plane orthogonal to primary diretion
    const real_t nextBoundary = rd[idx] > 0.0f ? firstBoundary + 1.0f : firstBoundary;
    real_t minDelta = (nextBoundary - currentPosition[idx]) / rd[idx];

    real_t intersectionLength = tmax - tmin;
    // first plane intersection may lie outside the bounding box
    if (intersectionLength < minDelta) {
        // use midpoint of entire ray intersection as a constant integration value
        updateTraverse<real_t, dim>(currentPosition, rd, intersectionLength * 0.5f);

        *sinogramPtr = weight * intersectionLength * tex<data_t, dim>(volume, currentPosition);
        return;
    }

    /**
     * otherwise first plane intersection inside bounding box
     * add first line segment and move to first interior point
     */
    updateTraverse<real_t, dim>(currentPosition, rd, minDelta * 0.5f);
    data_t pixelValue = weight * minDelta * tex<data_t, dim>(volume, currentPosition);

    // from here on use tmin as an indication of the current position along the ray
    tmin += minDelta;

    // if next point isn't last
    if (tmax - tmin > 1.0f) {
        updateTraverse<real_t, dim>(currentPosition, rd, (minDelta + 1.0f) * 0.5f);
        tmin += 1.0f;
        pixelValue += weight * tex<data_t, dim>(volume, currentPosition);

        // while interior intersection points remain
        while (tmax - tmin > 1.0f) {
            updateTraverse<real_t, dim>(currentPosition, rd, 1.0f);
            tmin += 1.0f;
            pixelValue += weight * tex<data_t, dim>(volume, currentPosition);
        }
    }

    updateTraverse<real_t, dim>(currentPosition, rd, (tmax - tmin + 1.0f) * 0.5f);
    pixelValue += weight * (tmax - tmin) * tex<data_t, dim>(volume, currentPosition);

    *sinogramPtr = pixelValue;
}

/// fetches double at position x, layer layer from a 1D layered texture
__device__ __forceinline__ double tex1DLayeredd(hipTextureObject_t texObj, const float x,
                                                const int layer)
{
    uint2 rt = tex1DLayered<uint2>(texObj, x, layer);
    return __hiloint2double(rt.y, rt.x);
}

/// template specialization for layered texture fetches
template <>
double tex1DLayered<double>(hipTextureObject_t texObj, elsa::real_t x, const int layer)
{
    x = x - 0.5f;

    elsa::real_t i = floor(x);

    elsa::real_t a = x - i;

    double T[2];
    T[0] = tex1DLayeredd(texObj, i, layer);
    T[1] = tex1DLayeredd(texObj, i + 1, layer);

    return (1 - a) * T[0] + a * T[1];
}

/// fetches double at position (x,y), layer layer from a 2D layered texture
__device__ __forceinline__ double tex2DLayeredd(hipTextureObject_t texObj, const float x,
                                                const float y, const int layer)
{
    uint2 rt = tex2DLayered<uint2>(texObj, x, y, layer);
    return __hiloint2double(rt.y, rt.x);
}

/// template specialization for layered texture fetches
template <>
double tex2DLayered<double>(hipTextureObject_t texObj, elsa::real_t x, elsa::real_t y,
                            const int layer)
{
    x = x - 0.5f;
    y = y - 0.5f;

    elsa::real_t i = floor(x);
    elsa::real_t j = floor(y);

    elsa::real_t a = x - i;
    elsa::real_t b = y - j;

    double T[2][2];
    T[0][0] = tex2DLayeredd(texObj, i, j, layer);
    T[1][0] = tex2DLayeredd(texObj, i + 1, j, layer);
    T[0][1] = tex2DLayeredd(texObj, i, j + 1, layer);
    T[1][1] = tex2DLayeredd(texObj, i + 1, j + 1, layer);

    return (1 - a) * (1 - b) * T[0][0] + a * (1 - b) * T[1][0] + (1 - a) * b * T[0][1]
           + a * b * T[1][1];
}

// TODO: check if sorting can be used to make this even faster
template <typename data_t, uint32_t dim>
__global__ void __launch_bounds__(elsa::TraverseJosephsCUDA<data_t, dim>::MAX_THREADS_PER_BLOCK)
    traverseAdjointFastKernel(int8_t* const __restrict__ volume, const uint64_t volumePitch,
                              hipTextureObject_t sinogram,
                              const int8_t* const __restrict__ rayOrigins,
                              const uint32_t originPitch, const int8_t* const __restrict__ proj,
                              const uint32_t projPitch, const uint32_t numAngles,
                              const uint32_t offset)
{

    using real_t = elsa::real_t;

    int x = blockIdx.x;
    int y = dim == 3 ? blockIdx.y : blockIdx.y * blockDim.x + threadIdx.x;
    int z = dim == 3 ? blockIdx.z * blockDim.x + threadIdx.x : 0;

    data_t& voxelRef =
        *(data_t*) (volume + x * sizeof(data_t) + y * volumePitch + z * volumePitch * gridDim.y);

    real_t voxelCenter[dim];
    voxelCenter[0] = x + 0.5f;
    if (dim == 3) {
        voxelCenter[dim - 2] = y + 0.5f;
        voxelCenter[dim - 1] = z + offset + 0.5f;
    } else {
        voxelCenter[1] = y + offset + 0.5f;
    }

    data_t val = 0.0f;
    for (uint i = 0; i < numAngles; i++) {
        const int8_t* const projPtr = proj + i * projPitch * dim;
        const real_t* const rayOrigin = (real_t*) (rayOrigins + i * originPitch);

        // compute ray direction
        real_t rd[dim];
#pragma unroll
        for (uint j = 0; j < dim; j++)
            rd[j] = voxelCenter[j] - rayOrigin[j];

        real_t pixelCoord[dim];

        gesqmv<real_t, dim>(projPtr, rd, pixelCoord, projPitch);

        // convert to homogenous coordinates
        pixelCoord[0] /= pixelCoord[dim - 1];

        if (dim == 3) {
            pixelCoord[1] /= pixelCoord[dim - 1];
            val += tex2DLayered<data_t>(sinogram, pixelCoord[0], pixelCoord[1], i);
        } else {
            val += tex1DLayered<data_t>(sinogram, pixelCoord[0], i);
        }
    }

    voxelRef = val;
}

/*
 * atomicAdd() for doubles is only supported on devices of compute capability 6.0 or higher
 * implementation taken straight from the CUDA C programming guide:
 * https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#atomic-functions
 */
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600
__device__ __forceinline__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull = (unsigned long long int*) address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val + __longlong_as_double(assumed)));

        // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

/// backprojects the weighted sinogram value to a given pixel
template <typename data_t, uint dim>
__device__ __forceinline__ void
    backproject2(int8_t* const __restrict__ volume, const uint64_t* const __restrict__ p,
                 uint32_t* const __restrict__ voxelCoord,
                 const elsa::real_t* const __restrict__ voxelCoordf,
                 const typename elsa::TraverseJosephsCUDA<data_t, dim>::BoundingBox& boxMax,
                 const elsa::real_t* const __restrict__ frac, const data_t weightedVal)
{

    data_t* volumeXPtr = (data_t*) (volume + p[0] * voxelCoord[0] + p[1] * voxelCoord[1]);
    data_t val = (1.0f - frac[1]) * weightedVal;
    atomicAdd(volumeXPtr, val);

    // volume[i,j+1]
    voxelCoord[1] = voxelCoord[1] < boxMax[1] - 1 ? voxelCoordf[1] + 1 : boxMax[1] - 1;
    volumeXPtr = (data_t*) (volume + p[0] * voxelCoord[0] + p[1] * voxelCoord[1]);
    val = frac[1] * weightedVal;
    atomicAdd(volumeXPtr, val);
}

/// backprojects the weighted sinogram value to a given voxel
template <typename data_t, uint dim>
__device__ __forceinline__ void
    backproject4(int8_t* const __restrict__ volume, const uint64_t* const __restrict__ p,
                 uint32_t* const __restrict__ voxelCoord,
                 const elsa::real_t* const __restrict__ voxelCoordf,
                 const typename elsa::TraverseJosephsCUDA<data_t, dim>::BoundingBox& boxMax,
                 const elsa::real_t* const __restrict__ frac, const data_t weightedVal)
{
    data_t* volumeXPtr =
        (data_t*) (volume + p[0] * voxelCoord[0] + p[1] * voxelCoord[1] + p[2] * voxelCoord[2]);
    data_t val = (1.0f - frac[1]) * (1.0f - frac[2]) * weightedVal;
    atomicAdd(volumeXPtr, val);
    // frac[0] is 0

    // volume[i,j+1,k]
    voxelCoord[1] = voxelCoord[1] < boxMax[1] - 1.0f ? voxelCoordf[1] + 1.0f : boxMax[1] - 1.0f;
    volumeXPtr =
        (data_t*) (volume + p[0] * voxelCoord[0] + p[1] * voxelCoord[1] + p[2] * voxelCoord[2]);
    val = frac[1] * (1.0f - frac[2]) * weightedVal;
    atomicAdd(volumeXPtr, val);

    // volume[i,j+1,k+1]
    voxelCoord[2] = voxelCoord[2] < boxMax[2] - 1.0f ? voxelCoordf[2] + 1.0f : boxMax[2] - 1.0f;
    volumeXPtr =
        (data_t*) (volume + p[0] * voxelCoord[0] + p[1] * voxelCoord[1] + p[2] * voxelCoord[2]);
    val = frac[1] * frac[2] * weightedVal;
    atomicAdd(volumeXPtr, val);

    // volume[i,j,k+1]
    voxelCoord[1] = voxelCoordf[1] < 0.0f ? 0 : voxelCoordf[1];
    volumeXPtr =
        (data_t*) (volume + p[0] * voxelCoord[0] + p[1] * voxelCoord[1] + p[2] * voxelCoord[2]);
    val = (1.0f - frac[1]) * frac[2] * weightedVal;
    atomicAdd(volumeXPtr, val);
}

/// convenience method for backprojecting to a given pixel/voxel
template <typename data_t, uint dim>
__device__ __forceinline__ void
    backproject(int8_t* const __restrict__ volume, const uint64_t* const __restrict__ p,
                uint32_t* const __restrict__ voxelCoord,
                const elsa::real_t* const __restrict__ voxelCoordf,
                const typename elsa::TraverseJosephsCUDA<data_t, dim>::BoundingBox& boxMax,
                const elsa::real_t* const __restrict__ frac, const data_t weightedVal)
{
    if (dim == 3)
        backproject4<data_t, dim>(volume, p, voxelCoord, voxelCoordf, boxMax, frac, weightedVal);
    else
        backproject2<data_t, dim>(volume, p, voxelCoord, voxelCoordf, boxMax, frac, weightedVal);
}

/// swaps the values of a and b
template <typename T>
__device__ __forceinline__ void swap(T& a, T& b)
{
    T c = a;
    a = b;
    b = c;
}

template <typename data_t, uint dim>
__global__ void __launch_bounds__(elsa::TraverseJosephsCUDA<data_t, dim>::MAX_THREADS_PER_BLOCK)
    traverseAdjointKernel(int8_t* const __restrict__ volume, const uint64_t volumePitch,
                          const int8_t* const __restrict__ sinogram, const uint64_t sinogramPitch,
                          const int8_t* const __restrict__ rayOrigins, const uint32_t originPitch,
                          const int8_t* const __restrict__ projInv, const uint32_t projPitch,
                          typename elsa::TraverseJosephsCUDA<data_t, dim>::BoundingBox boxMax)
{

    using real_t = elsa::real_t;

    const int8_t* const projInvPtr =
        dim == 3 ? projInv + (blockIdx.z * blockDim.x + threadIdx.x) * projPitch * 3
                 : projInv + (blockIdx.y * blockDim.x + threadIdx.x) * projPitch * 2;

    const real_t* const rayOrigin =
        dim == 3 ? (real_t*) (rayOrigins + (blockIdx.z * blockDim.x + threadIdx.x) * originPitch)
                 : (real_t*) (rayOrigins + (blockIdx.y * blockDim.x + threadIdx.x) * originPitch);

    const data_t sinogramVal =
        dim == 3 ? *((data_t*) (sinogram
                                + ((blockIdx.z * blockDim.x + threadIdx.x) * gridDim.y + blockIdx.y)
                                      * sinogramPitch)
                     + blockIdx.x)
                 : *((data_t*) (sinogram + (blockIdx.y * blockDim.x + threadIdx.x) * sinogramPitch)
                     + blockIdx.x);

    // homogenous pixel coordinates
    real_t pixelCoord[dim];
    pixelCoord[0] = blockIdx.x + 0.5f;
    pixelCoord[dim - 1] = 1.0f;
    if (dim == 3)
        pixelCoord[1] = blockIdx.y + 0.5f;

    // compute ray direction
    real_t rd[dim];
    gesqmv<real_t, dim>(projInvPtr, pixelCoord, rd, projPitch);
    normalize<real_t, dim>(rd);

    // find volume intersections
    real_t tmin, tmax;
    if (!box_intersect<real_t, dim>(rayOrigin, rd, boxMax.max, tmin, tmax))
        return;

    int stepDir[dim];
    real_t tdelta[dim];
    initStepDirection<real_t, dim>(rd, stepDir);
    initDelta<real_t, dim>(rd, stepDir, tdelta);

    real_t currentPosition[dim];
    pointAt<real_t, dim>(rayOrigin, rd, tmin, currentPosition);
    projectOntoBox<real_t, dim>(currentPosition, boxMax.max);

    uint32_t voxelCoord[dim];
    if (!closestVoxel<real_t, dim>(currentPosition, boxMax.max, voxelCoord, rd))
        return;

    // determine primary direction
    uint32_t idx = minIndex<real_t, dim>(tdelta);
    const int s = stepDir[idx];

    uint64_t permutation[dim];
    permutation[0] = sizeof(data_t);
    permutation[1] = volumePitch;
    if (dim == 3)
        permutation[dim - 1] = volumePitch * boxMax[1];

    // find distance to next plane orthogonal to primary diretion
    real_t nextBoundary = rd[idx] > 0.0f ? voxelCoord[idx] + 1 : voxelCoord[idx];
    real_t minDelta = (nextBoundary - currentPosition[idx]) / rd[idx];

    uint32_t entryDir = 0;
    for (uint i = 1; i < dim; i++)
        // current position is already projected onto the box, so a direct comparison works
        if (currentPosition[i] == 0 || currentPosition[i] == boxMax[i])
            entryDir = i;

    real_t intersectionLength = tmax - tmin;

    real_t voxelCoordf[dim], frac[dim];

    // subtract 0.5 from current position to get voxel coordinates
    for (uint i = 0; i < dim; i++) {
        currentPosition[i] -= 0.5f;
    }

    // permute indices, so that entry direction is at first index
    swap<real_t>(rd[0], rd[entryDir]);
    swap<real_t>(currentPosition[0], currentPosition[entryDir]);
    swap<uint32_t>(voxelCoord[0], voxelCoord[entryDir]);
    swap<real_t>(tdelta[0], tdelta[entryDir]);
    swap<real_t>(boxMax[0], boxMax[entryDir]);
    swap<uint64_t>(permutation[0], permutation[entryDir]);

    // first plane intersection may lie outside the bounding box
    if (intersectionLength < minDelta) {
        // use midpoint of entire ray intersection with bounding box as a constant integration value
        updateTraverse<real_t, dim>(currentPosition, rd, intersectionLength * 0.5f);
        for (uint i = 0; i < dim; i++) {
            voxelCoordf[i] = floorf(currentPosition[i]);
            frac[i] = currentPosition[i] - voxelCoordf[i];
            voxelCoord[i] = fmax(voxelCoordf[i], static_cast<real_t>(0));
        }
        backproject<data_t, dim>(volume, permutation, voxelCoord, voxelCoordf, boxMax, frac,
                                 intersectionLength * sinogramVal);
        return;
    }

    /**
     * otherwise first plane intersection inside bounding box
     * add first line segment and move to first interior point
     */
    updateTraverse<real_t, dim>(currentPosition, rd, minDelta * 0.5f);
    for (uint i = 0; i < dim; i++) {
        voxelCoordf[i] = floorf(currentPosition[i]);
        frac[i] = currentPosition[i] - voxelCoordf[i];
        voxelCoord[i] = fmax(voxelCoordf[i], static_cast<real_t>(0));
    }
    backproject<data_t, dim>(volume, permutation, voxelCoord, voxelCoordf, boxMax, frac,
                             minDelta * sinogramVal);
    // from here on use tmin as an indication of the current position along the ray
    tmin += minDelta;

    idx = minIndex<real_t, dim>(tdelta);
    // permute indices, so that primary direction is at first index
    swap<real_t>(rd[0], rd[idx]);
    swap<real_t>(currentPosition[0], currentPosition[idx]);
    swap<uint32_t>(voxelCoord[0], voxelCoord[idx]);
    swap<real_t>(tdelta[0], tdelta[idx]);
    swap<real_t>(boxMax[0], boxMax[idx]);
    swap<uint64_t>(permutation[0], permutation[idx]);

    // if next point isn't last
    if (tmax - tmin > tdelta[0]) {
        updateTraverse<real_t, dim>(currentPosition, rd, (minDelta + tdelta[0]) * 0.5f);
        minDelta = tdelta[0];
        tmin += minDelta;

        // set up values at idx manually, might lead to errors else
        currentPosition[0] = round(currentPosition[0]);
        frac[0] = 0.0f;
        voxelCoord[0] = (uint32_t) currentPosition[0];

        for (uint i = 1; i < dim; i++) {
            voxelCoordf[i] = floorf(currentPosition[i]);
            frac[i] = currentPosition[i] - voxelCoordf[i];
            voxelCoord[i] = fmax(static_cast<real_t>(0), voxelCoordf[i]);
        }
        backproject<data_t, dim>(volume, permutation, voxelCoord, voxelCoordf, boxMax, frac,
                                 minDelta * sinogramVal);

        // while interior intersection points remain
        while (tmin + minDelta < tmax) {
            updateTraverse<real_t, dim>(currentPosition, rd, minDelta);
            tmin += minDelta;

            voxelCoord[0] += s;
            for (uint i = 1; i < dim; i++) {
                voxelCoordf[i] = floorf(currentPosition[i]);
                frac[i] = currentPosition[i] - voxelCoordf[i];
                voxelCoord[i] = fmax(voxelCoordf[i], static_cast<real_t>(0));
            }
            backproject<data_t, dim>(volume, permutation, voxelCoord, voxelCoordf, boxMax, frac,
                                     minDelta * sinogramVal);
        }
    }

    updateTraverse<real_t, dim>(currentPosition, rd, (tmax + minDelta - tmin) * 0.5f);
    for (uint32_t i = 1; i < dim; i++) {
        // for large volumes numerical errors sometimes cause currentPosition of the last voxel
        // to lie outside boxMax although ideally it should not even exceed boxMax-0.5; currently
        // solved by readjusting the coordinates if needed
        // TODO: try updating the traversal using pointAt() instead
        voxelCoordf[i] = floorf(currentPosition[i]);
        frac[i] = currentPosition[i] - voxelCoordf[i];
        voxelCoord[i] = fmax(voxelCoordf[i], static_cast<real_t>(0));

        if (voxelCoord[i] >= boxMax[i]) {
            voxelCoord[i] = boxMax[i] - 1.0f;
            frac[i] = 0.5f;
        }
    }

    real_t mainDirPos = currentPosition[0];
    for (uint32_t i = 0; i < dim; i++) {
        // move to exit point
        currentPosition[i] = currentPosition[i] + 0.5f + rd[i] * (tmax - tmin) * 0.5f;

        // distance to border
        currentPosition[i] = fabs(fmin(currentPosition[i], boxMax[i] - currentPosition[i]));
    }

    // find direction closest to border
    uint32_t exitDir = minIndex<real_t, dim>(currentPosition);

    if (exitDir == 0) {
        // again handle this case manually
        voxelCoordf[0] = (real_t) voxelCoord[0] + s;
        voxelCoord[0] =
            fmin(fmax(voxelCoordf[0], static_cast<real_t>(0)), boxMax[0] - static_cast<real_t>(1));
        frac[0] = mainDirPos - voxelCoordf[0];
    } else {
        voxelCoordf[0] = floorf(mainDirPos);
        frac[0] = mainDirPos - voxelCoordf[0];
        voxelCoord[0] = fmax(static_cast<real_t>(0), voxelCoordf[0]);
    }

    // permute indices, so that exit direction is at first index
    swap<uint32_t>(voxelCoord[0], voxelCoord[exitDir]);
    swap<real_t>(voxelCoordf[0], voxelCoordf[exitDir]);
    swap<real_t>(frac[0], frac[exitDir]);
    swap<real_t>(boxMax[0], boxMax[exitDir]);
    swap<uint64_t>(permutation[0], permutation[exitDir]);
    backproject<data_t, dim>(volume, permutation, voxelCoord, voxelCoordf, boxMax, frac,
                             (tmax - tmin) * sinogramVal);
}

namespace elsa
{

    template <typename data_t, uint32_t dim>
    void TraverseJosephsCUDA<data_t, dim>::traverseForward(
        const dim3 blocks, const int threads, hipTextureObject_t volume,
        int8_t* const __restrict__ sinogram, const uint64_t sinogramPitch,
        const int8_t* const __restrict__ rayOrigins, const uint32_t originPitch,
        const int8_t* const __restrict__ projInv, const uint32_t projPitch,
        const BoundingBox& boxMax, const hipStream_t stream)
    {
        traverseForwardKernel<data_t, dim><<<blocks, threads, 0, stream>>>(
            volume, sinogram, sinogramPitch, rayOrigins, originPitch, projInv, projPitch, boxMax);
    }

    template <typename data_t, uint32_t dim>
    void TraverseJosephsCUDA<data_t, dim>::traverseAdjoint(
        const dim3 blocks, const int threads, int8_t* const __restrict__ volume,
        const uint64_t volumePitch, const int8_t* const __restrict__ sinogram,
        const uint64_t sinogramPitch, const int8_t* const __restrict__ rayOrigins,
        const uint32_t originPitch, const int8_t* const __restrict__ projInv,
        const uint32_t projPitch, const BoundingBox& boxMax, const hipStream_t stream)
    {
        traverseAdjointKernel<data_t, dim>
            <<<blocks, threads, 0, stream>>>(volume, volumePitch, sinogram, sinogramPitch,
                                             rayOrigins, originPitch, projInv, projPitch, boxMax);
    }

    template <typename data_t, uint32_t dim>
    void TraverseJosephsCUDA<data_t, dim>::traverseAdjointFast(
        const dim3 blocks, const int threads, int8_t* const __restrict__ volume,
        const uint64_t volumePitch, hipTextureObject_t sinogram,
        const int8_t* const __restrict__ rayOrigins, const uint32_t originPitch,
        const int8_t* const __restrict__ proj, const uint32_t projPitch, const uint32_t numAngles,
        const uint32_t zOffset, const hipStream_t stream)
    {
        traverseAdjointFastKernel<data_t, dim>
            <<<blocks, threads, 0, stream>>>(volume, volumePitch, sinogram, rayOrigins, originPitch,
                                             proj, projPitch, numAngles, zOffset);
    }

    // template instantiations
    template struct TraverseJosephsCUDA<float, 2>;
    template struct TraverseJosephsCUDA<float, 3>;

    template struct TraverseJosephsCUDA<double, 2>;
    template struct TraverseJosephsCUDA<double, 3>;
} // namespace elsa