#include "hip/hip_runtime.h"
#include "TraverseJosephsCUDA.cuh"
#include "SharedArray.cuh"
#include "Matrix.cuh"
#include "Geometry.cuh"
#include "TraversalUtils.cuh"
#include "Tex.cuh"
#include "AtomicAdd.cuh"

#include <type_traits>

constexpr uint32_t MAX_THREADS_PER_BLOCK =
    elsa::TraverseJosephsCUDA<float, 3>::MAX_THREADS_PER_BLOCK;

template <typename data_t, uint32_t size>
using EasyAccessSharedArray =
    elsa::detail::EasyAccessSharedArray<data_t, size, MAX_THREADS_PER_BLOCK>;

/// determines the voxel that contains a point, if the point is on a border the voxel in the ray
/// direction is favored
template <typename real_t, uint32_t dim>
__device__ __forceinline__ bool closestVoxel(const EasyAccessSharedArray<real_t, dim>& point,
                                             const EasyAccessSharedArray<real_t, dim>& boxMax,
                                             EasyAccessSharedArray<uint32_t, dim>& voxelCoord,
                                             const EasyAccessSharedArray<real_t, dim>& rd)
{
#pragma unroll
    for (uint32_t i = 0; i < dim; i++) {
        // point has been projected onto box => point[i]>=0, can use uint32_t
        uint32_t fl = trunc(point[i]);
        // for Joseph's also consider rays running along the "left" boundary
        voxelCoord[i] = fl == point[i] && rd[i] <= 0.0f && point[i] != 0.0f ? fl - 1 : fl;
        if (voxelCoord[i] >= boxMax[i])
            return false;
    }
    return true;
}

template <typename real_t, uint32_t dim>
__device__ __forceinline__ void updateTraverse(EasyAccessSharedArray<real_t, dim>& p,
                                               const EasyAccessSharedArray<real_t, dim>& rd,
                                               const real_t dist)
{
#pragma unroll
    for (uint32_t i = 0; i < dim; i++)
        p[i] += rd[i] * dist;
}

template <typename data_t, uint32_t dim>
__global__ void __launch_bounds__(elsa::TraverseJosephsCUDA<data_t, dim>::MAX_THREADS_PER_BLOCK)
    traverseForwardKernel(
        hipTextureObject_t volume, int8_t* const __restrict__ sinogram,
        const uint64_t sinogramPitch, const uint32_t sinogramOffsetX,
        const int8_t* const __restrict__ rayOrigins, const uint32_t originPitch,
        const int8_t* const __restrict__ projInv, const uint32_t projPitch,
        const typename elsa::TraverseJosephsCUDA<data_t, dim>::BoundingBox boundingBox)
{

    using real_t = elsa::real_t;

    const int8_t* const projInvPtr = projInv + blockIdx.x * projPitch * dim;

    const real_t* const rayOrigin = (real_t*) (rayOrigins + blockIdx.x * originPitch);

    const uint32_t xCoord = sinogramOffsetX + blockDim.x * blockIdx.z + threadIdx.x;

    data_t* sinogramPtr =
        ((data_t*) (sinogram + (blockIdx.x * gridDim.y + blockIdx.y) * sinogramPitch) + xCoord);

    *sinogramPtr = 0;

    // homogenous pixel coordinates
    real_t pixelCoord[dim];
    pixelCoord[0] = xCoord + 0.5f;
    pixelCoord[dim - 1] = 1.0f;
    if (dim == 3)
        pixelCoord[1] = blockIdx.y + 0.5f;

    __shared__ real_t currentPositionsShared[dim * MAX_THREADS_PER_BLOCK];
    __shared__ real_t rayDirectionsShared[dim * MAX_THREADS_PER_BLOCK];
    __shared__ real_t boxMaxsShared[dim * MAX_THREADS_PER_BLOCK];

    EasyAccessSharedArray<real_t, dim> boxMax{boxMaxsShared};
#pragma unroll
    for (uint32_t i = 0; i < dim; ++i)
        boxMax[i] = boundingBox[i];

    // compute ray direction
    EasyAccessSharedArray<real_t, dim> rd{rayDirectionsShared};
    gesqmv<real_t, dim>(projInvPtr, pixelCoord, rd, projPitch);

    // determine main direction
    const uint32_t idx = maxAbsIndex<real_t, dim>(rd);
    const real_t rdMax = abs(rd[idx]);

    real_t rn = rnorm<real_t, dim>(rd);

    real_t weight = rn / rdMax;

// normalize ray direction to have length 1/-1 in the main direction
#pragma unroll
    for (uint32_t i = 0; i < dim; ++i)
        rd[i] /= rdMax;

    // find volume intersections
    real_t tmin, tmax;
    if (!box_intersect<real_t, dim>(rayOrigin, rd, boxMax, tmin, tmax))
        return;

    EasyAccessSharedArray<real_t, dim> currentPosition{currentPositionsShared};
    pointAt<real_t, dim>(rayOrigin, rd, tmin, currentPosition);

    // truncate as currentPosition is non-negative
    const real_t fl = trunc(currentPosition[idx]);
    // for Joseph's also consider rays running along the "left" boundary
    const real_t firstBoundary = fl == currentPosition[idx] && rd[idx] < 0.0f ? fl - 1.0f : fl;

    // find distance to next plane orthogonal to primary diretion
    const real_t nextBoundary = rd[idx] > 0.0f ? firstBoundary + 1.0f : firstBoundary;
    real_t minDelta = (nextBoundary - currentPosition[idx]) / rd[idx];

    real_t intersectionLength = tmax - tmin;
    // first plane intersection may lie outside the bounding box
    if (intersectionLength < minDelta) {
        // use midpoint of entire ray intersection as a constant integration value
        updateTraverse<real_t, dim>(currentPosition, rd, intersectionLength * 0.5f);

        *sinogramPtr = weight * intersectionLength * tex<data_t, dim>(volume, currentPosition);
        return;
    }

    /**
     * otherwise first plane intersection inside bounding box
     * add first line segment and move to first interior point
     */
    updateTraverse<real_t, dim>(currentPosition, rd, minDelta * 0.5f);
    data_t pixelValue = weight * minDelta * tex<data_t, dim>(volume, currentPosition);

    // from here on use tmin as an indication of the current position along the ray
    tmin += minDelta;

    // if next point isn't last
    if (tmax - tmin > 1.0f) {
        updateTraverse<real_t, dim>(currentPosition, rd, (minDelta + 1.0f) * 0.5f);
        tmin += 1.0f;
        pixelValue += weight * tex<data_t, dim>(volume, currentPosition);

        // while interior intersection points remain
        while (tmax - tmin > 1.0f) {
            updateTraverse<real_t, dim>(currentPosition, rd, 1.0f);
            tmin += 1.0f;
            pixelValue += weight * tex<data_t, dim>(volume, currentPosition);
        }
    }

    updateTraverse<real_t, dim>(currentPosition, rd, (tmax - tmin + 1.0f) * 0.5f);
    pixelValue += weight * (tmax - tmin) * tex<data_t, dim>(volume, currentPosition);

    *sinogramPtr = pixelValue;
}

/// fetches double at position x, layer layer from a 1D layered texture
__device__ __forceinline__ double tex1DLayeredd(hipTextureObject_t texObj, const float x,
                                                const int layer)
{
    uint2 rt = tex1DLayered<uint2>(texObj, x, layer);
    return __hiloint2double(rt.y, rt.x);
}

/// template specialization for layered texture fetches
template <>
double tex1DLayered<double>(hipTextureObject_t texObj, float x, const int layer)
{
    x = x - 0.5f;

    float i = floorf(x);

    double a = x - i;

    double T[2];
    T[0] = tex1DLayeredd(texObj, i, layer);
    T[1] = tex1DLayeredd(texObj, i + 1, layer);

    return (1 - a) * T[0] + a * T[1];
}

/// fetches double at position (x,y), layer layer from a 2D layered texture
__device__ __forceinline__ double tex2DLayeredd(hipTextureObject_t texObj, const float x,
                                                const float y, const int layer)
{
    uint2 rt = tex2DLayered<uint2>(texObj, x, y, layer);
    return __hiloint2double(rt.y, rt.x);
}

/// template specialization for layered texture fetches
template <>
double tex2DLayered<double>(hipTextureObject_t texObj, float x, float y, const int layer)
{
    x = x - 0.5f;
    y = y - 0.5f;

    float i = floorf(x);
    float j = floorf(y);

    double a = x - i;
    double b = y - j;

    double T[2][2];
    T[0][0] = tex2DLayeredd(texObj, i, j, layer);
    T[1][0] = tex2DLayeredd(texObj, i + 1, j, layer);
    T[0][1] = tex2DLayeredd(texObj, i, j + 1, layer);
    T[1][1] = tex2DLayeredd(texObj, i + 1, j + 1, layer);

    return (1 - a) * (1 - b) * T[0][0] + a * (1 - b) * T[1][0] + (1 - a) * b * T[0][1]
           + a * b * T[1][1];
}

// TODO: check if sorting can be used to make this even faster
template <typename data_t, uint32_t dim>
__global__ void __launch_bounds__(MAX_THREADS_PER_BLOCK)
    traverseAdjointFastKernel(int8_t* const __restrict__ volume, const uint64_t volumePitch,
                              const uint32_t volumeOffsetX, const uint32_t steps,
                              hipTextureObject_t sinogram,
                              const int8_t* const __restrict__ rayOrigins,
                              const uint32_t originPitch, const int8_t* const __restrict__ proj,
                              const uint32_t projPitch, const uint32_t numAngles)
{

    using real_t = elsa::real_t;

    int x = volumeOffsetX + threadIdx.x;
    int y = blockIdx.y;
    int z = blockIdx.x;

    real_t voxelCenter[dim];
    voxelCenter[0] = x + 0.5f;
    voxelCenter[1] = y + 0.5f;
    if (dim == 3)
        voxelCenter[dim - 1] = z + 0.5f;

    extern __shared__ real_t valuesShared[];

    EasyAccessSharedArray<real_t, 1> values{valuesShared};
    for (uint32_t step = 0; step < steps; ++step)
        values[step] = 0;

    for (uint i = 0; i < numAngles; i++) {
        const int8_t* const projPtr = proj + i * projPitch * dim;
        const real_t* const rayOrigin = (real_t*) (rayOrigins + i * originPitch);

        voxelCenter[0] = x + 0.5f;

        real_t rd[dim];
#pragma unroll
        for (uint j = 0; j < dim; j++)
            rd[j] = voxelCenter[j] - rayOrigin[j];

        // compute ray direction
        for (uint32_t step = 0; step < steps; ++step) {

            real_t pixelCoord[dim];
            gesqmv<real_t, dim>(projPtr, rd, pixelCoord, projPitch);

            // convert to homogenous coordinates
            pixelCoord[0] /= pixelCoord[dim - 1];

            if (dim == 3) {
                pixelCoord[1] /= pixelCoord[dim - 1];
                values[step] += tex2DLayered<data_t>(sinogram, pixelCoord[0], pixelCoord[1], i);
            } else {
                values[step] += tex1DLayered<data_t>(sinogram, pixelCoord[0], i);
            }

            voxelCenter[0] += blockDim.x;
            rd[0] += blockDim.x;
        }
    }

    for (uint32_t step = 0; step < steps; ++step) {
        int x = volumeOffsetX + step * blockDim.x + threadIdx.x;
        data_t& voxelRef = *(data_t*) (volume + x * sizeof(data_t) + y * volumePitch
                                       + z * volumePitch * gridDim.y);
        voxelRef = values[step];
    }
}

/// backprojects the weighted sinogram value to a given pixel
template <typename data_t, uint dim>
__device__ __forceinline__ void
    backproject2(int8_t* const __restrict__ volume, const EasyAccessSharedArray<uint64_t, dim>& p,
                 EasyAccessSharedArray<uint32_t, dim>& voxelCoord,
                 const EasyAccessSharedArray<elsa::real_t, dim>& voxelCoordf,
                 const EasyAccessSharedArray<elsa::real_t, dim>& boxMax,
                 const EasyAccessSharedArray<elsa::real_t, dim>& frac, const data_t weightedVal)
{

    data_t* volumeXPtr = (data_t*) (volume + p[0] * voxelCoord[0] + p[1] * voxelCoord[1]);
    data_t val = (1.0f - frac[1]) * weightedVal;
    atomicAdd(volumeXPtr, val);

    // volume[i,j+1]
    voxelCoord[1] = voxelCoord[1] < boxMax[1] - 1 ? voxelCoordf[1] + 1 : boxMax[1] - 1;
    volumeXPtr = (data_t*) (volume + p[0] * voxelCoord[0] + p[1] * voxelCoord[1]);
    val = frac[1] * weightedVal;
    atomicAdd(volumeXPtr, val);
}

/// backprojects the weighted sinogram value to a given voxel
template <typename data_t, uint dim>
__device__ __forceinline__ void
    backproject4(int8_t* const __restrict__ volume, const EasyAccessSharedArray<uint64_t, dim>& p,
                 EasyAccessSharedArray<uint32_t, dim>& voxelCoord,
                 const EasyAccessSharedArray<elsa::real_t, dim>& voxelCoordf,
                 const EasyAccessSharedArray<elsa::real_t, dim>& boxMax,
                 const EasyAccessSharedArray<elsa::real_t, dim>& frac, const data_t weightedVal)
{
    data_t* volumeXPtr =
        (data_t*) (volume + p[0] * voxelCoord[0] + p[1] * voxelCoord[1] + p[2] * voxelCoord[2]);
    data_t val = (1.0f - frac[1]) * (1.0f - frac[2]) * weightedVal;
    atomicAdd(volumeXPtr, val);
    // frac[0] is 0

    // volume[i,j+1,k]
    voxelCoord[1] = voxelCoord[1] < boxMax[1] - 1.0f ? voxelCoordf[1] + 1.0f : boxMax[1] - 1.0f;
    volumeXPtr =
        (data_t*) (volume + p[0] * voxelCoord[0] + p[1] * voxelCoord[1] + p[2] * voxelCoord[2]);
    val = frac[1] * (1.0f - frac[2]) * weightedVal;
    atomicAdd(volumeXPtr, val);

    // volume[i,j+1,k+1]
    voxelCoord[2] = voxelCoord[2] < boxMax[2] - 1.0f ? voxelCoordf[2] + 1.0f : boxMax[2] - 1.0f;
    volumeXPtr =
        (data_t*) (volume + p[0] * voxelCoord[0] + p[1] * voxelCoord[1] + p[2] * voxelCoord[2]);
    val = frac[1] * frac[2] * weightedVal;
    atomicAdd(volumeXPtr, val);

    // volume[i,j,k+1]
    voxelCoord[1] = voxelCoordf[1] < 0.0f ? 0 : voxelCoordf[1];
    volumeXPtr =
        (data_t*) (volume + p[0] * voxelCoord[0] + p[1] * voxelCoord[1] + p[2] * voxelCoord[2]);
    val = (1.0f - frac[1]) * frac[2] * weightedVal;
    atomicAdd(volumeXPtr, val);
}

/// convenience method for backprojecting to a given pixel/voxel
template <typename data_t, uint dim>
__device__ __forceinline__ void
    backproject(int8_t* const __restrict__ volume, const EasyAccessSharedArray<uint64_t, dim>& p,
                EasyAccessSharedArray<uint32_t, dim>& voxelCoord,
                const EasyAccessSharedArray<elsa::real_t, dim>& voxelCoordf,
                const EasyAccessSharedArray<elsa::real_t, dim>& boxMax,
                const EasyAccessSharedArray<elsa::real_t, dim>& frac, const data_t weightedVal)
{
    if (dim == 3)
        backproject4<data_t, dim>(volume, p, voxelCoord, voxelCoordf, boxMax, frac, weightedVal);
    else
        backproject2<data_t, dim>(volume, p, voxelCoord, voxelCoordf, boxMax, frac, weightedVal);
}

/// swaps the values of a and b
template <typename T>
__device__ __forceinline__ void swap(T& a, T& b)
{
    T c = a;
    a = b;
    b = c;
}

template <typename data_t, uint dim>
__global__ void __launch_bounds__(elsa::TraverseJosephsCUDA<data_t, dim>::MAX_THREADS_PER_BLOCK)
    traverseAdjointKernel(int8_t* const __restrict__ volume, const uint64_t volumePitch,
                          const int8_t* const __restrict__ sinogram, const uint64_t sinogramPitch,
                          const uint32_t sinogramOffsetX,
                          const int8_t* const __restrict__ rayOrigins, const uint32_t originPitch,
                          const int8_t* const __restrict__ projInv, const uint32_t projPitch,
                          typename elsa::TraverseJosephsCUDA<data_t, dim>::BoundingBox boundingBox)
{

    using real_t = elsa::real_t;

    const int8_t* const projInvPtr = projInv + blockIdx.x * projPitch * dim;

    const real_t* const rayOrigin = (real_t*) (rayOrigins + blockIdx.x * originPitch);

    const uint32_t xCoord = sinogramOffsetX + blockDim.x * blockIdx.z + threadIdx.x;

    data_t& sinogramVal =
        *((data_t*) (sinogram + (blockIdx.x * gridDim.y + blockIdx.y) * sinogramPitch) + xCoord);

    // homogenous pixel coordinates
    real_t pixelCoord[dim];
    pixelCoord[0] = xCoord + 0.5f;
    pixelCoord[dim - 1] = 1.0f;
    if (dim == 3)
        pixelCoord[1] = blockIdx.y + 0.5f;

    __shared__ real_t currentPositionsShared[dim * MAX_THREADS_PER_BLOCK];
    __shared__ real_t rayDirectionsShared[dim * MAX_THREADS_PER_BLOCK];
    __shared__ uint32_t voxelCoordsShared[dim * MAX_THREADS_PER_BLOCK];
    __shared__ real_t voxelCoordfsShared[dim * MAX_THREADS_PER_BLOCK];
    __shared__ real_t fracsShared[dim * MAX_THREADS_PER_BLOCK];
    __shared__ real_t tdeltasShared[dim * MAX_THREADS_PER_BLOCK];
    __shared__ uint64_t permutationsShared[dim * MAX_THREADS_PER_BLOCK];
    __shared__ real_t boxMaxsShared[dim * MAX_THREADS_PER_BLOCK];

    EasyAccessSharedArray<real_t, dim> boxMax{boxMaxsShared};
#pragma unroll
    for (uint32_t i = 0; i < dim; ++i)
        boxMax[i] = boundingBox[i];

    // compute ray direction
    EasyAccessSharedArray<real_t, dim> rd{rayDirectionsShared};
    gesqmv<real_t, dim>(projInvPtr, pixelCoord, rd, projPitch);
    normalize<real_t, dim>(rd);

    // find volume intersections
    real_t tmin, tmax;
    if (!box_intersect<real_t, dim>(rayOrigin, rd, boxMax, tmin, tmax))
        return;

    EasyAccessSharedArray<real_t, dim> tdelta{tdeltasShared};
    initDelta<real_t, dim>(rd, tdelta);

    EasyAccessSharedArray<real_t, dim> currentPosition{currentPositionsShared};
    pointAt<real_t, dim>(rayOrigin, rd, tmin, currentPosition);
    projectOntoBox<real_t, dim>(currentPosition, boxMax);

    EasyAccessSharedArray<uint32_t, dim> voxelCoord{voxelCoordsShared};
    if (!closestVoxel<real_t, dim>(currentPosition, boxMax, voxelCoord, rd))
        return;

    // determine primary direction
    uint32_t idx = minIndex<real_t, dim>(tdelta);
    const int s = ((rd[idx] > 0.0f) - (rd[idx] < 0.0f));

    EasyAccessSharedArray<uint64_t, dim> permutation{permutationsShared};
    permutation[0] = sizeof(data_t);
    permutation[1] = volumePitch;
    if (dim == 3)
        permutation[dim - 1] = volumePitch * boxMax[1];

    // find distance to next plane orthogonal to primary diretion
    real_t nextBoundary = rd[idx] > 0.0f ? voxelCoord[idx] + 1 : voxelCoord[idx];
    real_t minDelta = (nextBoundary - currentPosition[idx]) / rd[idx];

    uint32_t entryDir = 0;
    for (uint i = 1; i < dim; i++)
        // current position is already projected onto the box, so a direct comparison works
        if (currentPosition[i] == 0 || currentPosition[i] == boxMax[i])
            entryDir = i;

    real_t intersectionLength = tmax - tmin;

    EasyAccessSharedArray<real_t, dim> voxelCoordf{voxelCoordfsShared};
    EasyAccessSharedArray<real_t, dim> frac{fracsShared};

    // subtract 0.5 from current position to get voxel coordinates
    for (uint i = 0; i < dim; i++) {
        currentPosition[i] -= 0.5f;
    }

    // permute indices, so that entry direction is at first index
    swap<real_t>(rd[0], rd[entryDir]);
    swap<real_t>(currentPosition[0], currentPosition[entryDir]);
    swap<uint32_t>(voxelCoord[0], voxelCoord[entryDir]);
    swap<real_t>(tdelta[0], tdelta[entryDir]);
    swap<real_t>(boxMax[0], boxMax[entryDir]);
    swap<uint64_t>(permutation[0], permutation[entryDir]);

    // first plane intersection may lie outside the bounding box
    if (intersectionLength < minDelta) {
        // use midpoint of entire ray intersection with bounding box as a constant integration value
        updateTraverse<real_t, dim>(currentPosition, rd, intersectionLength * 0.5f);
        for (uint i = 0; i < dim; i++) {
            voxelCoordf[i] = floorf(currentPosition[i]);
            frac[i] = currentPosition[i] - voxelCoordf[i];
            voxelCoord[i] = fmax(voxelCoordf[i], static_cast<real_t>(0));
        }
        backproject<data_t, dim>(volume, permutation, voxelCoord, voxelCoordf, boxMax, frac,
                                 intersectionLength * sinogramVal);
        return;
    }

    /**
     * otherwise first plane intersection inside bounding box
     * add first line segment and move to first interior point
     */
    updateTraverse<real_t, dim>(currentPosition, rd, minDelta * 0.5f);
    for (uint i = 0; i < dim; i++) {
        voxelCoordf[i] = floorf(currentPosition[i]);
        frac[i] = currentPosition[i] - voxelCoordf[i];
        voxelCoord[i] = fmax(voxelCoordf[i], static_cast<real_t>(0));
    }
    backproject<data_t, dim>(volume, permutation, voxelCoord, voxelCoordf, boxMax, frac,
                             minDelta * sinogramVal);
    // from here on use tmin as an indication of the current position along the ray
    tmin += minDelta;

    idx = minIndex<real_t, dim>(tdelta);
    // permute indices, so that primary direction is at first index
    swap<real_t>(rd[0], rd[idx]);
    swap<real_t>(currentPosition[0], currentPosition[idx]);
    swap<uint32_t>(voxelCoord[0], voxelCoord[idx]);
    swap<real_t>(tdelta[0], tdelta[idx]);
    swap<real_t>(boxMax[0], boxMax[idx]);
    swap<uint64_t>(permutation[0], permutation[idx]);

    // if next point isn't last
    if (tmax - tmin > tdelta[0]) {
        updateTraverse<real_t, dim>(currentPosition, rd, (minDelta + tdelta[0]) * 0.5f);
        minDelta = tdelta[0];
        tmin += minDelta;

        // set up values at idx manually, might lead to errors else
        currentPosition[0] = round(currentPosition[0]);
        frac[0] = 0.0f;
        voxelCoord[0] = (uint32_t) currentPosition[0];

        for (uint i = 1; i < dim; i++) {
            voxelCoordf[i] = floorf(currentPosition[i]);
            frac[i] = currentPosition[i] - voxelCoordf[i];
            voxelCoord[i] = fmax(static_cast<real_t>(0), voxelCoordf[i]);
        }
        backproject<data_t, dim>(volume, permutation, voxelCoord, voxelCoordf, boxMax, frac,
                                 minDelta * sinogramVal);

        // while interior intersection points remain
        while (tmin + minDelta < tmax) {
            updateTraverse<real_t, dim>(currentPosition, rd, minDelta);
            tmin += minDelta;

            voxelCoord[0] += s;
            for (uint i = 1; i < dim; i++) {
                voxelCoordf[i] = floorf(currentPosition[i]);
                frac[i] = currentPosition[i] - voxelCoordf[i];
                voxelCoord[i] = fmax(voxelCoordf[i], static_cast<real_t>(0));
            }
            backproject<data_t, dim>(volume, permutation, voxelCoord, voxelCoordf, boxMax, frac,
                                     minDelta * sinogramVal);
        }
    }

    updateTraverse<real_t, dim>(currentPosition, rd, (tmax + minDelta - tmin) * 0.5f);
    for (uint32_t i = 1; i < dim; i++) {
        // for large volumes numerical errors sometimes cause currentPosition of the last voxel
        // to lie outside boxMax although ideally it should not even exceed boxMax-0.5; currently
        // solved by readjusting the coordinates if needed
        // TODO: try updating the traversal using pointAt() instead
        voxelCoordf[i] = floorf(currentPosition[i]);
        frac[i] = currentPosition[i] - voxelCoordf[i];
        voxelCoord[i] = fmax(voxelCoordf[i], static_cast<real_t>(0));

        if (voxelCoord[i] >= boxMax[i]) {
            voxelCoord[i] = boxMax[i] - 1.0f;
            frac[i] = 0.5f;
        }
    }

    real_t mainDirPos = currentPosition[0];
    for (uint32_t i = 0; i < dim; i++) {
        // move to exit point
        currentPosition[i] = currentPosition[i] + 0.5f + rd[i] * (tmax - tmin) * 0.5f;

        // distance to border
        currentPosition[i] = fabs(fmin(currentPosition[i], boxMax[i] - currentPosition[i]));
    }

    // find direction closest to border
    uint32_t exitDir = minIndex<real_t, dim>(currentPosition);

    if (exitDir == 0) {
        // again handle this case manually
        voxelCoordf[0] = (real_t) voxelCoord[0] + s;
        voxelCoord[0] =
            fmin(fmax(voxelCoordf[0], static_cast<real_t>(0)), boxMax[0] - static_cast<real_t>(1));
        frac[0] = mainDirPos - voxelCoordf[0];
    } else {
        voxelCoordf[0] = floorf(mainDirPos);
        frac[0] = mainDirPos - voxelCoordf[0];
        voxelCoord[0] = fmax(static_cast<real_t>(0), voxelCoordf[0]);
    }

    // permute indices, so that exit direction is at first index
    swap<uint32_t>(voxelCoord[0], voxelCoord[exitDir]);
    swap<real_t>(voxelCoordf[0], voxelCoordf[exitDir]);
    swap<real_t>(frac[0], frac[exitDir]);
    swap<real_t>(boxMax[0], boxMax[exitDir]);
    swap<uint64_t>(permutation[0], permutation[exitDir]);
    backproject<data_t, dim>(volume, permutation, voxelCoord, voxelCoordf, boxMax, frac,
                             (tmax - tmin) * sinogramVal);
}

namespace elsa
{

    template <typename data_t, uint32_t dim>
    void TraverseJosephsCUDA<data_t, dim>::traverseForward(
        dim3 sinogramDims, int threads, hipTextureObject_t volume, int8_t* __restrict__ sinogram,
        uint64_t sinogramPitch, const int8_t* __restrict__ rayOrigins, uint32_t originPitch,
        const int8_t* __restrict__ projInv, uint32_t projPitch, const BoundingBox& boxMax)
    {
        uint32_t numImageBlocks = sinogramDims.z / threads;
        uint32_t remaining = sinogramDims.z % threads;
        uint32_t offset = numImageBlocks * threads;

        if (numImageBlocks > 0) {
            dim3 grid(sinogramDims.x, sinogramDims.y, numImageBlocks);
            traverseForwardKernel<data_t, dim><<<grid, threads>>>(volume, sinogram, sinogramPitch,
                                                                  0, rayOrigins, originPitch,
                                                                  projInv, projPitch, boxMax);
        }

        if (remaining > 0) {
            hipStream_t remStream;

            if (hipStreamCreate(&remStream) != hipSuccess)
                throw std::logic_error(
                    "TraverseJosephsCUDA: Couldn't create stream for remaining images");

            dim3 grid(sinogramDims.x, sinogramDims.y, 1);
            traverseForwardKernel<data_t, dim><<<grid, remaining, 0, remStream>>>(
                volume, sinogram, sinogramPitch, offset, rayOrigins, originPitch, projInv,
                projPitch, boxMax);

            if (hipStreamDestroy(remStream) != hipSuccess)
                throw std::logic_error("TraverseJosephsCUDA: Couldn't destroy cudaStream object");
        }
    }

    template <typename data_t, uint32_t dim>
    void TraverseJosephsCUDA<data_t, dim>::traverseAdjoint(
        dim3 sinogramDims, int threads, int8_t* __restrict__ volume, uint64_t volumePitch,
        const int8_t* __restrict__ sinogram, uint64_t sinogramPitch,
        const int8_t* __restrict__ rayOrigins, uint32_t originPitch,
        const int8_t* __restrict__ projInv, uint32_t projPitch, const BoundingBox& boxMax)
    {
        uint32_t numImageBlocks = sinogramDims.z / threads;
        uint32_t remaining = sinogramDims.z % threads;
        uint32_t offset = numImageBlocks * threads;

        if (numImageBlocks > 0) {
            dim3 grid(sinogramDims.x, sinogramDims.y, numImageBlocks);
            traverseAdjointKernel<data_t, dim>
                <<<grid, threads>>>(volume, volumePitch, sinogram, sinogramPitch, 0, rayOrigins,
                                    originPitch, projInv, projPitch, boxMax);
        }

        if (remaining > 0) {
            hipStream_t remStream;

            if (hipStreamCreate(&remStream) != hipSuccess)
                throw std::logic_error(
                    "TraverseJosephsCUDA: Couldn't create stream for remaining images");

            dim3 grid(sinogramDims.x, sinogramDims.y, 1);
            traverseAdjointKernel<data_t, dim><<<grid, remaining, 0, remStream>>>(
                volume, volumePitch, sinogram, sinogramPitch, offset, rayOrigins, originPitch,
                projInv, projPitch, boxMax);

            if (hipStreamDestroy(remStream) != hipSuccess)
                throw std::logic_error("TraverseJosephsCUDA: Couldn't destroy cudaStream object");
        }
    }

    template <typename data_t, uint32_t dim>
    void TraverseJosephsCUDA<data_t, dim>::traverseAdjointFast(
        dim3 volumeDims, int threads, int8_t* __restrict__ volume, uint64_t volumePitch,
        hipTextureObject_t sinogram, const int8_t* __restrict__ rayOrigins, uint32_t originPitch,
        const int8_t* __restrict__ proj, uint32_t projPitch, uint32_t numAngles)
    {
        uint32_t numImageBlocks = volumeDims.z / threads;
        uint32_t remaining = volumeDims.z % threads;
        uint32_t offset = numImageBlocks * threads;

        if (numImageBlocks > 0) {
            dim3 grid(volumeDims.x, volumeDims.y, 1);
            traverseAdjointFastKernel<data_t, dim>
                <<<grid, threads, numImageBlocks * MAX_THREADS_PER_BLOCK * sizeof(data_t)>>>(
                    volume, volumePitch, 0, numImageBlocks, sinogram, rayOrigins, originPitch, proj,
                    projPitch, numAngles);
        }

        if (remaining > 0) {
            hipStream_t remStream;

            if (hipStreamCreate(&remStream) != hipSuccess)
                throw std::logic_error(
                    "TraverseJosephsCUDA: Couldn't create stream for remaining images");

            dim3 grid(volumeDims.x, volumeDims.y, 1);
            traverseAdjointFastKernel<data_t, dim>
                <<<grid, threads, remaining * sizeof(data_t), remStream>>>(
                    volume, volumePitch, offset, 1, sinogram, rayOrigins, originPitch, proj,
                    projPitch, numAngles);

            if (hipStreamDestroy(remStream) != hipSuccess)
                throw std::logic_error("TraverseJosephsCUDA: Couldn't destroy cudaStream object");
        }
    }

    // template instantiations
    template struct TraverseJosephsCUDA<float, 2>;
    template struct TraverseJosephsCUDA<float, 3>;

    template struct TraverseJosephsCUDA<double, 2>;
    template struct TraverseJosephsCUDA<double, 3>;
} // namespace elsa
