#include "CudnnContext.h"

namespace elsa
{
    namespace ml
    {
        namespace detail
        {
            CudnnContext::CudnnContext()
            {
                hipblasCreate(&cublasHandle_);
                hipdnnCreate(&cudnnHandle_);
            }

            CudnnContext::~CudnnContext()
            {
                hipblasDestroy(cublasHandle_);
                hipdnnDestroy(cudnnHandle_);
            }

            hipblasHandle_t& CudnnContext::getCublasHandle() { return cublasHandle_; }
            const hipblasHandle_t& CudnnContext::getCublasHandle() const { return cublasHandle_; }

            hipdnnHandle_t& CudnnContext::getCudnnHandle() { return cudnnHandle_; }
            const hipdnnHandle_t& CudnnContext::getCudnnHandle() const { return cudnnHandle_; }

            const float CudnnContext::One = 1.f;
            const float CudnnContext::Zero = 0.f;
            const float CudnnContext::MinusOne = -1.f;

        } // namespace detail
    }     // namespace ml
} // namespace elsa